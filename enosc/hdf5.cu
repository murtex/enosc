#include "hip/hip_runtime.h"
/**
 * hdf5.cu
 * 20150704
 *
 * hdf5 observer
 */

	/* includes */
#include "hdf5.h"

#include <xis/singleton.h>
#include <xis/logger.h>

#include <typeinfo>

	/* con/destruction */
enosc::HDF5::HDF5()
{

		/* default configuration */
	_deflate = 0;

}

enosc::HDF5::~HDF5()
{
}

	/* configuration */
void enosc::HDF5::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* base call */
	enosc::Observer::configure( config, groupname );

		/* parse group settings */
	std::string settingname = groupname + "/deflate";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _deflate );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "deflate: " << _deflate << "\n";

}

	/* observation */
void enosc::HDF5::init( enosc::Ensemble const & ensemble, enosc::Stepper const & stepper, std::string const & filename )
{

		/* base call */
	enosc::Observer::init( ensemble, stepper, filename );

		/* create file */
	_file = H5::H5File( filename.c_str(), H5F_ACC_TRUNC );

		/* set datatype */
	if ( typeid( enosc::scalar ) == typeid( float ) )
		_datatype = H5::PredType::NATIVE_FLOAT;
	else if ( typeid( enosc::scalar ) == typeid( double ) )
		_datatype = H5::PredType::NATIVE_DOUBLE;
	else
		throw std::runtime_error( "invalid type: enosc::HDF5::init, enosc::scalar" );

		/* create static datasets */
	H5::DataSet dataset = _file.createDataSet( "size", _datatype, H5::DataSpace() );
	dataset.write( &_size, H5::PredType::NATIVE_UINT );

	dataset = _file.createDataSet( "dim", _datatype, H5::DataSpace() );
	unsigned int dim = ensemble.get_dim();
	dataset.write( &dim, H5::PredType::NATIVE_UINT );

	hsize_t dims = ensemble.get_epsilons().size();
	dataset = _file.createDataSet( "epsilons", _datatype, H5::DataSpace( 1, &dims ) );
	enosc::device_vector const & epsilons = ensemble.get_epsilons();
	dataset.write( enosc::host_vector( epsilons.begin(), epsilons.end() ).data(), _datatype );

	dims = ensemble.get_betas().size();
	dataset = _file.createDataSet( "betas", _datatype, H5::DataSpace( 1, &dims ) );
	enosc::device_vector const & betas = ensemble.get_betas();
	dataset.write( enosc::host_vector( betas.begin(), betas.end() ).data(), _datatype );

	dims = stepper.get_times().size();
	dataset = _file.createDataSet( "times", _datatype, H5::DataSpace( 1, &dims ) );
	dataset.write( stepper.get_times().data(), _datatype );

		/* initialize dynamic datasets */
	H5::Group group = _file.createGroup( "raw" );

	H5::DSetCreatPropList props; /* oscillators */
	hsize_t chunks[5] = {1, ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), _size};
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	hsize_t mdims[5] = {stepper.get_times().size(), ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), _size};
	_raw_x = group.createDataSet( "x", _datatype, H5::DataSpace( 5, mdims ), props );
	_raw_dxdt = group.createDataSet( "dxdt", _datatype, H5::DataSpace( 5, mdims ), props );

	chunks[4] = _meanfield ? 1 : 0; /* meanfield */
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	mdims[4] = _meanfield ? 1 : 0;
	_raw_mf = group.createDataSet( "mf", _datatype, H5::DataSpace( 5, mdims ), props );
	_raw_dmfdt = group.createDataSet( "dmfdt", _datatype, H5::DataSpace( 5, mdims ), props );

}

void enosc::HDF5::observe( enosc::Ensemble & ensemble, unsigned int step, enosc::scalar time )
{

		/* write oscillators */
	hsize_t dims_in[4] = {ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), ensemble.get_size()};
	H5::DataSpace dataspace_in( 4, dims_in );
	hsize_t starts_in[4] = {0, 0, 0, 0};
	hsize_t counts_in[4] = {ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), _size};
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	H5::DataSpace dataspace_out( _raw_x.getSpace() );
	hsize_t starts_out[5] = {step, 0, 0, 0, 0};
	hsize_t counts_out[5] = {1, ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), _size};
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	enosc::device_vector const & state = ensemble.get_state();
	_raw_x.write( enosc::host_vector( state.begin(), state.end() ).data(), _datatype, dataspace_in, dataspace_out );

    enosc::device_vector const & deriv = ensemble.compute_deriv( state, time );
    _raw_dxdt.write( enosc::host_vector( deriv.begin(), deriv.end() ).data(), _datatype, dataspace_in, dataspace_out );

		/* write meanfield */
	dims_in[3] = 1;
	dataspace_in = H5::DataSpace( 4, dims_in );
	counts_in[3] = _meanfield ? 1 : 0;
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	dataspace_out = _raw_mf.getSpace();
	counts_out[4] = _meanfield ? 1 : 0;
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	enosc::device_vector const & mean = ensemble.compute_mean( state );
	_raw_mf.write( enosc::host_vector( mean.begin(), mean.end() ).data(), _datatype, dataspace_in, dataspace_out );

	enosc::device_vector const & meanderiv = ensemble.compute_mean( deriv );
	_raw_dmfdt.write( enosc::host_vector( meanderiv.begin(), meanderiv.end() ).data(), _datatype, dataspace_in, dataspace_out );

}

