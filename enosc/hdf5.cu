#include "hip/hip_runtime.h"
/**
 * hdf5.cu
 * 20150704
 *
 * hdf5 observer
 */

	/* includes */
#include "hdf5.h"

#include <xis/singleton.h>
#include <xis/logger.h>

#include <typeinfo>

	/* con/destruction */
enosc::HDF5::HDF5()
{

		/* default configuration */
	_deflate = 0;

}

enosc::HDF5::~HDF5()
{
}

	/* configuration */
void enosc::HDF5::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* base call */
	enosc::Observer::configure( config, groupname );

		/* parse group settings */
	std::string settingname = groupname + "/deflate";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _deflate );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "deflate: " << _deflate << "\n";

}

	/* observation */
void enosc::HDF5::init( enosc::Ensemble const & ensemble, enosc::Stepper const & stepper, std::string const & filename )
{

		/* base call */
	enosc::Observer::init( ensemble, stepper, filename );

		/* create file */
	_file = H5::H5File( filename.c_str(), H5F_ACC_TRUNC );

		/* set datatype */
	if ( typeid( enosc::scalar ) == typeid( float ) )
		_datatype = H5::PredType::NATIVE_FLOAT;
	else if ( typeid( enosc::scalar ) == typeid( double ) )
		_datatype = H5::PredType::NATIVE_DOUBLE;
	else
		throw std::runtime_error( "invalid type (enosc::HDF5::init): enosc::scalar" );

		/* create static datasets */
	enosc::host_vector const & ctimes = stepper.get_times(); /* times */
	enosc::host_vector times( ctimes.begin() + _transition, ctimes.end() );
	hsize_t dim = times.size();
	H5::DataSet dataset = _file.createDataSet( "times", _datatype, H5::DataSpace( 1, &dim ) );
	dataset.write( times.data(), _datatype );

	enosc::scalar dt = stepper.get_dt(); /* dt */
	dataset = _file.createDataSet( "dt", _datatype, H5::DataSpace() );
	dataset.write( &dt, _datatype );

	enosc::device_vector const & epsilons = ensemble.get_epsilons(); /* epsilons */
	dim = epsilons.size();
	dataset = _file.createDataSet( "epsilons", _datatype, H5::DataSpace( 1, &dim ) );
	dataset.write( enosc::host_vector( epsilons.begin(), epsilons.end() ).data(), _datatype );

	enosc::device_vector const & betas = ensemble.get_betas(); /* betas */
	dim = betas.size();
	dataset = _file.createDataSet( "betas", _datatype, H5::DataSpace( 1, &dim ) );
	dataset.write( enosc::host_vector( betas.begin(), betas.end() ).data(), _datatype );

		/* initialize dynamic datasets */

			/* raw */
	H5::Group group = _file.createGroup( "raw" );

	H5::DSetCreatPropList props; /* oscillators */
	hsize_t chunks[5] = {1, ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), (hsize_t) (_oscillator ? 1 : 0)};
	chunks[4] = (_oscillator ? 1 : 0) * (_track_raw ? 1 : 0);
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	hsize_t dims[5] = {times.size(), ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), (hsize_t) (_oscillator ? 1 : 0)};
	dims[4] = (_oscillator ? 1 : 0) * (_track_raw ? 1 : 0);
	_raw_x = group.createDataSet( "x", _datatype, H5::DataSpace( 5, dims ), props );
	_raw_dxdt = group.createDataSet( "dxdt", _datatype, H5::DataSpace( 5, dims ), props );

	chunks[4] = (_ensemble ? 1 : 0) * (_track_raw ? 1 : 0); /* mean */
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	dims[4] = (_ensemble ? 1 : 0) * (_track_raw ? 1 : 0);
	_raw_mx = group.createDataSet( "mx", _datatype, H5::DataSpace( 5, dims ), props );
	_raw_dmxdt = group.createDataSet( "dmxdt", _datatype, H5::DataSpace( 5, dims ), props );

			/* polar */
	group = _file.createGroup( "polar" );

	chunks[1] = 2; /* oscillators */
	chunks[4] = (_oscillator ? 1 : 0) * (_track_polar ? 1 : 0);
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	dims[1] = 2;
	dims[4] = (_oscillator ? 1 : 0) * (_track_polar ? 1 : 0);
	_polar_x = group.createDataSet( "x", _datatype, H5::DataSpace( 5, dims ), props );
	_polar_dxdt = group.createDataSet( "dxdt", _datatype, H5::DataSpace( 5, dims ), props );

	chunks[4] = (_ensemble ? 1 : 0) * (_track_polar ? 1 : 0); /* mean */
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	dims[4] = (_ensemble ? 1 : 0) * (_track_polar ? 1 : 0);
	_polar_mx = group.createDataSet( "mx", _datatype, H5::DataSpace( 5, dims ), props );
	_polar_dmxdt = group.createDataSet( "dmxdt", _datatype, H5::DataSpace( 5, dims ), props );

	chunks[4] = (_meanfield ? 1 : 0) * (_track_polar ? 1 : 0); /* meanfield */
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	dims[4] = (_meanfield ? 1 : 0) * (_track_polar ? 1 : 0);
	_polar_mf = group.createDataSet( "mf", _datatype, H5::DataSpace( 5, dims ), props );
	_polar_dmfdt = group.createDataSet( "dmfdt", _datatype, H5::DataSpace( 5, dims ), props );

			/* funnel */
	group = _file.createGroup( "funnel" );

	chunks[1] = 1; /* mean */
	chunks[4] = (_ensemble ? 1 : 0) * (_track_funnel ? 1 : 0);
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	dims[1] = 1;
	dims[4] = (_ensemble ? 1 : 0) * (_track_funnel ? 1 : 0);
	_funnel_mx = group.createDataSet( "mx", _datatype, H5::DataSpace( 5, dims ), props );

	chunks[4] = (_meanfield ? 1 : 0) * (_track_funnel ? 1 : 0); /* meanfield */
	if ( _deflate != 0 ) {
		props.setChunk( 5, chunks );
		props.setDeflate( _deflate );
		props.setShuffle();
	}

	dims[4] = (_meanfield ? 1 : 0) * (_track_funnel ? 1 : 0);
	_funnel_mf = group.createDataSet( "mf", _datatype, H5::DataSpace( 5, dims ), props );

}

void enosc::HDF5::observe( enosc::Ensemble & ensemble, unsigned int step, enosc::scalar time )
{

		/* prepare buffers */
	enosc::device_vector & state = ensemble.get_state();
	enosc::device_vector const & state_next = ensemble.get_state_next();
	
	enosc::device_vector & polar = ensemble.get_polar();
	enosc::device_vector & deriv = ensemble.get_deriv();

	enosc::device_vector & mean = ensemble.get_mean();

		/* update ensemble center (transition phase) */
	if ( step < _transition ) {
		if ( _centering ) {
			ensemble.compute_mean( state );

			thrust::transform(
				_center.begin(), _center.end(),
				thrust::make_transform_iterator(
					mean.begin(), thrust::placeholders::_1 / _transition ),
				_center.begin(), thrust::plus< enosc::scalar >() );
		}

		return;
	}

		/* write raw oscillators */
	hsize_t dims_in[4] = {ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), ensemble.get_size()};
	hsize_t starts_in[4] = {0, 0, 0, 0};
	hsize_t counts_in[4] = {ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), (hsize_t) (_oscillator ? 1 : 0) * (_track_raw ? 1 : 0)};
	H5::DataSpace dataspace_in = H5::DataSpace( 4, dims_in );
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	hsize_t starts_out[5] = {step - _transition, 0, 0, 0, 0};
	hsize_t counts_out[5] = {1, ensemble.get_dim(), ensemble.get_epsilons().size(), ensemble.get_betas().size(), (hsize_t) (_oscillator ? 1 : 0) * (_track_raw ? 1 : 0)};
	H5::DataSpace dataspace_out = _raw_x.getSpace();
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	_raw_x.write( enosc::host_vector( state.begin(), state.end() ).data(), _datatype, dataspace_in, dataspace_out );

	thrust::transform( /* compute derivative */
		state_next.begin(), state_next.end(),
		state.begin(),
		deriv.begin(), thrust::minus< enosc::scalar >() );

	_raw_dxdt.write( enosc::host_vector( deriv.begin(), deriv.end() ).data(), _datatype, dataspace_in, dataspace_out );

		/* write polar oscillators */
	dims_in[0] = 2;
	counts_in[0] = 2;
	counts_in[3] = (_oscillator ? 1 : 0) * (_track_polar ? 1 : 0);
	dataspace_in = H5::DataSpace( 4, dims_in );
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	counts_out[1] = 2;
	counts_out[4] = (_oscillator ? 1 : 0) * (_track_polar ? 1 : 0);
	dataspace_out = _polar_x.getSpace();
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	if ( _centering ) /* center ensemble */
		thrust::transform(
			state.begin(), state.end(),
			thrust::make_permutation_iterator(
				_center.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 / ensemble.get_size() ) ),
	        state.begin(), thrust::minus< enosc::scalar >() );

	ensemble.compute_polar( state, deriv );
	_polar_x.write( enosc::host_vector( polar.begin(), polar.end() ).data(), _datatype, dataspace_in, dataspace_out );
	_polar_dxdt.write( enosc::host_vector( deriv.begin(), deriv.end() ).data(), _datatype, dataspace_in, dataspace_out );

		/* write mean funnel */
	dims_in[0] = 1;
	dims_in[3] = 1;
	counts_in[0] = 1;
	counts_in[3] = (_ensemble ? 1 : 0) * (_track_funnel ? 1 : 0);
	dataspace_in = H5::DataSpace( 4, dims_in );
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	counts_out[1] = 1;
	counts_out[4] = (_ensemble ? 1 : 0) * (_track_funnel ? 1 : 0);
	dataspace_out = _funnel_mx.getSpace();
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	compute_funnel( deriv, ensemble.get_size() );
	_funnel_mx.write( enosc::host_vector( _funnel.begin(), _funnel.end() ).data(), _datatype, dataspace_in, dataspace_out );

		/* write polar mean */
	dims_in[0] = 2;
	counts_in[0] = 2;
	counts_in[3] = (_ensemble ? 1 : 0) * (_track_polar ? 1 : 0);
	dataspace_in = H5::DataSpace( 4, dims_in );
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	counts_out[1] = 2;
	counts_out[4] = (_ensemble ? 1 : 0) * (_track_polar ? 1 : 0);
	dataspace_out = _polar_mx.getSpace();
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	ensemble.compute_mean( polar );
	_polar_mx.write( enosc::host_vector( mean.begin(), mean.end() ).data(), _datatype, dataspace_in, dataspace_out );

	ensemble.compute_mean( deriv );
	_polar_dmxdt.write( enosc::host_vector( mean.begin(), mean.end() ).data(), _datatype, dataspace_in, dataspace_out );

		/* write raw mean (raw meanfield) */
	dims_in[0] = ensemble.get_dim();
	counts_in[0] = ensemble.get_dim();
	counts_in[3] = (_ensemble ? 1 : 0) * (_track_raw ? 1 : 0);
	dataspace_in = H5::DataSpace( 4, dims_in );
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	counts_out[1] = ensemble.get_dim();
	counts_out[4] = (_ensemble ? 1 : 0) * (_track_raw ? 1 : 0);
	dataspace_out = _raw_mx.getSpace();
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	if ( _centering ) /* un-center ensemble */
		thrust::transform(
			state.begin(), state.end(),
			thrust::make_permutation_iterator(
				_center.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 / ensemble.get_size() ) ),
	        state.begin(), thrust::plus< enosc::scalar >() );

	ensemble.compute_mean( state );
	_raw_mx.write( enosc::host_vector( mean.begin(), mean.end() ).data(), _datatype, dataspace_in, dataspace_out );
	thrust::copy( mean.begin(), mean.end(), _tmp.begin() ); /* keep a backup of raw-mx */

	thrust::transform( /* re-compute derivative */
		state_next.begin(), state_next.end(),
		state.begin(),
		deriv.begin(), thrust::minus< enosc::scalar >() );

	ensemble.compute_mean( deriv );
	_raw_dmxdt.write( enosc::host_vector( mean.begin(), mean.end() ).data(), _datatype, dataspace_in, dataspace_out );

		/* write polar meanfield */
	dims_in[0] = 2;
	counts_in[0] = 2;
	counts_in[3] = (_meanfield ? 1 : 0) * (_track_polar ? 1 : 0);
	dataspace_in = H5::DataSpace( 4, dims_in );
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	counts_out[1] = 2;
	counts_out[4] = (_meanfield ? 1 : 0) * (_track_polar ? 1 : 0);
	dataspace_out = _polar_mf.getSpace();
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	if ( _centering ) /* re-center meanfield */
		thrust::transform(
			_tmp.begin(), _tmp.end(),
			_center.begin(),
	        _tmp.begin(), thrust::minus< enosc::scalar >() );

	ensemble.compute_polar( _tmp, mean ); /* use backup of raw-mx */
	_polar_mf.write( enosc::host_vector( polar.begin(), polar.end() ).data(), _datatype, dataspace_in, dataspace_out );
	_polar_dmfdt.write( enosc::host_vector( deriv.begin(), deriv.end() ).data(), _datatype, dataspace_in, dataspace_out );

		/* write meanfield funnel */
	dims_in[0] = 1;
	dims_in[3] = 1;
	counts_in[0] = 1;
	counts_in[3] = (_meanfield ? 1 : 0) * (_track_funnel ? 1 : 0);
	dataspace_in = H5::DataSpace( 4, dims_in );
	dataspace_in.selectHyperslab( H5S_SELECT_SET, counts_in, starts_in );

	counts_out[1] = 1;
	counts_out[4] = (_meanfield ? 1 : 0) * (_track_funnel ? 1 : 0);
	dataspace_out = _funnel_mf.getSpace();
	dataspace_out.selectHyperslab( H5S_SELECT_SET, counts_out, starts_out );

	compute_funnel( deriv, 1 );
	_funnel_mf.write( enosc::host_vector( _funnel.begin(), _funnel.end() ).data(), _datatype, dataspace_in, dataspace_out );

}

