#include "hip/hip_runtime.h"
/**
 * ensemble.cu
 * 20150703
 *
 * ensemble interface
 */

	/* includes */
#include "ensemble.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* con/destruction */
enosc::Ensemble::Ensemble()
{

		/* initialize configuration */
	_size = 0;
	_dim = 0;

}

	/* configuration */
void enosc::Ensemble::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/size";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _size );

	std::string paramname = groupname + "/epsilon";
	std::string stepname = groupname + "/epsilon_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_epsilons.resize( steps+1 );
		_epsilons[0] = start;
		for ( unsigned int i = 1; i < steps+1; ++i )
			_epsilons[i] = start + i * (stop-start) / (enosc::scalar) steps;

	}

	paramname = groupname + "/beta";
	stepname = groupname + "/beta_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_betas.resize( steps+1 );
		_betas[0] = start;
		for ( unsigned int i = 1; i < steps+1; ++i )
			_betas[i] = start + i * (stop-start) / (enosc::scalar) steps;

	}

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "size: " << _size << "\n";
	logger.log() << "dim: " << _dim << "\n";

	logger.log() << "epsilons: " << _epsilons << "\n";
	logger.log() << "betas: " << _betas << "\n";

}

	/* phase space */
void enosc::Ensemble::init( unsigned int seed, bool det, bool stoch )
{

		/* safeguard */
	if ( !det && !stoch )
		throw std::runtime_error( "invalid values: enosc::Ensemble::init, det | stoch" );

		/* prepare phase space buffers */
	_state.resize( _dim * _size * _epsilons.size() * _betas.size() ); /* phase state */

	if ( det ) /* derivatives */
		_deriv_det.resize( _state.size() );
	if ( stoch )
		_deriv_stoch.resize( _state.size() );

		/* prepare computation buffers */
	_meanfield.resize( _dim * _epsilons.size() * _betas.size() ); /* meanfield */

		/* initialize randomness */
	srand( seed );

		/* logging */
	size_t cuda_free;
	size_t cuda_total;

	hipMemGetInfo( &cuda_free, &cuda_total );

	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "cuda: " << ((cuda_total-cuda_free)>>20) << "/" << (cuda_total>>20) << "MiB\n";

}

	/* computation */
enosc::device_vector const & enosc::Ensemble::compute_deriv( enosc::device_vector const & state, enosc::scalar time )
{

		/* return pure deterministic/stochastic derivative */
	if ( _deriv_det.size() == 0 )
		return compute_deriv_stoch( state, time );

	else if ( _deriv_stoch.size() == 0 )
		return compute_deriv_det( state, time );

		/* return overall derivative */
	enosc::device_vector const & deriv_det = compute_deriv_det( state, time );
	enosc::device_vector const & deriv_stoch = compute_deriv_stoch( state, time );

	thrust::transform(
		deriv_det.begin(), deriv_det.end(),
		deriv_stoch.begin(),
		_deriv_det.begin(),
		thrust::plus< enosc::scalar >() );

	return _deriv_det;
}

