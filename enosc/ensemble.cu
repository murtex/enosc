#include "hip/hip_runtime.h"
/**
 * ensemble.cu
 * 20150703
 *
 * ensemble interface
 */

	/* includes */
#include "ensemble.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* con/destruction */
enosc::Ensemble::Ensemble()
{

		/* initialize configuration */
	_size = 0;
	_dim = 0;

}

	/* configuration */
void enosc::Ensemble::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/size";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _size );

	std::string paramname = groupname + "/epsilon";
	std::string stepname = groupname + "/epsilon_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_epsilons.resize( steps+1 );
		_epsilons[0] = start;
		for ( unsigned int i = 1; i < steps+1; ++i )
			_epsilons[i] = start + i * (stop-start) / (enosc::scalar) steps;

	}

	paramname = groupname + "/beta";
	stepname = groupname + "/beta_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_betas.resize( steps+1 );
		_betas[0] = start;
		for ( unsigned int i = 1; i < steps+1; ++i )
			_betas[i] = start + i * (stop-start) / (enosc::scalar) steps;

	}

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "size: " << _size << "\n";
	logger.log() << "dim: " << _dim << "\n";

	logger.log() << "epsilons: " << _epsilons << "\n";
	logger.log() << "betas: " << _betas << "\n";

}

	/* phase space */
void enosc::Ensemble::init( unsigned int seed, bool det, bool stoch )
{

		/* safeguard */
	if ( !det && !stoch )
		throw std::runtime_error( "invalid values: enosc::Ensemble::init, det | stoch" );

	if ( _dim < 2 || _epsilons.size() == 0 || _betas.size() == 0 || _size == 0 )
		throw std::runtime_error( "invalid values: enosc::Ensemble::init, _dim | _epsilons | _betas | _size" );

		/* prepare buffers */
	_state.resize( _dim * _epsilons.size() * _betas.size() * _size ); /* phase state */

	if ( det ) /* derivatives */
		_deriv_det.resize( _state.size() );
	if ( stoch )
		_deriv_stoch.resize( _state.size() );

	_mean.resize( _dim * _epsilons.size() * _betas.size() ); /* ensemble mean */

		/* initialize randomness */
	srand( seed );

		/* logging */
	size_t cuda_free;
	size_t cuda_total;

	hipMemGetInfo( &cuda_free, &cuda_total );

	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "cuda: " << ((cuda_total-cuda_free)>>20) << "/" << (cuda_total>>20) << "MiB\n";

}

	/* computation */
enosc::device_vector const & enosc::Ensemble::compute_deriv( enosc::device_vector const & state, enosc::scalar time )
{

		/* safeguard */
	if ( state.size() != _state.size() )
		throw std::runtime_error( "invalid value: enosc::Ensemble::compute_deriv, state" );

		/* return pure deterministic/stochastic derivative */
	if ( _deriv_det.size() == 0 )
		return compute_deriv_stoch( state, time );

	else if ( _deriv_stoch.size() == 0 )
		return compute_deriv_det( state, time );

		/* return composite derivative */
	compute_deriv_det( state, time );
	compute_deriv_stoch( state, time );

	thrust::transform(
		_deriv_det.begin(), _deriv_det.end(), /* summands input */
		_deriv_stoch.begin(),
		_deriv_det.begin(), /* sum output */
		thrust::plus< enosc::scalar >() );

	return _deriv_det;
}

enosc::device_vector const & enosc::Ensemble::compute_mean( enosc::device_vector const & buf )
{

		/* safeguard */
	if ( buf.size() != _state.size() )
		throw std::runtime_error( "invalid value: enosc::Ensemble::compute_mean, buf" );

		/* average ensemble */
	thrust::reduce_by_key(

		thrust::make_transform_iterator( /* keys */
			thrust::counting_iterator< unsigned int >( 0 ),
			thrust::placeholders::_1 / _size ),
		thrust::make_transform_iterator(
			thrust::counting_iterator< unsigned int >( 0 ),
			thrust::placeholders::_1 / _size ) + buf.size(),

		thrust::make_transform_iterator( /* scaled input */
			buf.begin(), thrust::placeholders::_1 / _size ),

		thrust::make_discard_iterator(), _mean.begin() ); /* keys, mean output */

	return _mean;
}

