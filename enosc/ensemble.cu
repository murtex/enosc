#include "hip/hip_runtime.h"
/**
 * ensemble.cu
 * 20150703
 *
 * ensemble interface
 */

	/* includes */
#include "ensemble.h"

#include <xis/singleton.h>
#include <xis/logger.h>

#include <enosc/kernels.h>

	/* con/destruction */
enosc::Ensemble::Ensemble()
{

		/* initialize configuration */
	_size = 0;
	_dim = 0;

}

	/* configuration */
void enosc::Ensemble::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/seed";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _seed );

	settingname = groupname + "/size";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _size );

	std::string paramname = groupname + "/epsilon";
	std::string stepname = groupname + "/epsilon_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_epsilons.resize( steps+1 );
		_epsilons[0] = start;
		for ( unsigned int i = 1; i < steps; ++i )
			_epsilons[i] = start + i * (stop-start) / (enosc::scalar) steps;
		if ( steps >= 1 )
			_epsilons[steps] = stop;

	}

	paramname = groupname + "/beta";
	stepname = groupname + "/beta_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_betas.resize( steps+1 );
		_betas[0] = start;
		for ( unsigned int i = 1; i < steps; ++i )
			_betas[i] = start + i * (stop-start) / (enosc::scalar) steps;
		if ( steps >= 1 )
			_betas[steps] = stop;

	}

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "seed: " << _seed << "\n";

	logger.log() << "size: " << _size << "\n";
	logger.log() << "dim: " << _dim << "\n";
	logger.log() << "polar: " << _fpolar << "\n";

	logger.log() << "epsilons: " << _epsilons << "\n";
	logger.log() << "betas: " << _betas << "\n";

}

	/* phase space */
void enosc::Ensemble::init()
{

		/* safeguard */
	if ( _dim < 2 || _size == 0 )
		throw std::runtime_error( "invalid values (enosc::Ensemble::init): _dim | _size" );

	if ( _epsilons.size() == 0 || _betas.size() == 0 )
		throw std::runtime_error( "invalid values (enosc::Enesemble::init): _epsilons | _betas" );

		/* prepare buffers */
	_state.resize( _dim * _epsilons.size() * _betas.size() * _size ); /* double buffered state */
	_state_next.resize( _state.size() );

	_polar.resize( 2 * _epsilons.size() * _betas.size() * _size ); /* polar transform */
	_deriv.resize( _state.size() ); /* derivative */

	_mean.resize( _dim * _epsilons.size() * _betas.size() ); /* ensemble mean */

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	size_t cuda_free;
	size_t cuda_total;
	hipMemGetInfo( &cuda_free, &cuda_total );
	logger.log() << "cuda: " << ((cuda_total-cuda_free)>>20) << "/" << (cuda_total>>20) << "MiB\n";

}

void enosc::Ensemble::swap()
{

		/* swap state buffers */
	_state = _state_next;

}

void enosc::Ensemble::compute_polar( enosc::device_vector const & buf, enosc::device_vector const & buf_deriv )
{

		/* safeguard */
	if ( buf.size() % (_dim * _epsilons.size() * _betas.size()) != 0 ||
			buf_deriv.size() % (_dim * _epsilons.size() * _betas.size()) != 0 )
		throw std::runtime_error( "invalid arguments (enosc::Ensemble::compute_polar): buf | buf_deriv" );

		/* compute polar transform */
	unsigned int size = buf.size() / (_dim * _epsilons.size() * _betas.size()); /* input ensemble size */
	if ( buf_deriv.size() < buf.size() )
		size = buf_deriv.size() / (_dim * _epsilons.size() * _betas.size());

	if ( _fpolar ) /* identity */
		thrust::for_each_n(
			thrust::make_zip_iterator( thrust::make_tuple(

				buf.begin(), /* polar input */
				buf.begin() + _epsilons.size() * _betas.size() * size,
				buf_deriv.begin(),
				buf_deriv.begin() + _epsilons.size() * _betas.size() * size,

				_polar.begin(), /* polar output */
				_polar.begin() + _epsilons.size() * _betas.size() * size,
				_deriv.begin(),
				_deriv.begin() + _epsilons.size() * _betas.size() * size ) ),

			_epsilons.size() * _betas.size() * size, enosc::PolarToPolarFull() );

	else /* cartesian to polar */
		thrust::for_each_n(
			thrust::make_zip_iterator( thrust::make_tuple(

				buf.begin(), /* cartesian input */
				buf.begin() + _epsilons.size() * _betas.size() * size,
				buf_deriv.begin(),
				buf_deriv.begin() + _epsilons.size() * _betas.size() * size,

				_polar.begin(), /* polar output */
				_polar.begin() + _epsilons.size() * _betas.size() * size,
				_deriv.begin(),
				_deriv.begin() + _epsilons.size() * _betas.size() * size ) ),

			_epsilons.size() * _betas.size() * size, enosc::CartesianToPolarFull() );

}

void enosc::Ensemble::compute_mean( enosc::device_vector const & buf )
{

		/* safeguard */
	if ( buf.size() % (_epsilons.size() * _betas.size() * _size) != 0 )
		throw std::runtime_error( "invalid argument (enosc::Ensemble::compute_mean): buf" );

		/* compute ensemble mean */
	thrust::reduce_by_key(

		thrust::make_transform_iterator( /* input keys */
			thrust::counting_iterator< unsigned int >( 0 ),
			thrust::placeholders::_1 / _size ),
		thrust::make_transform_iterator(
			thrust::counting_iterator< unsigned int >( 0 ),
			thrust::placeholders::_1 / _size ) + buf.size(),

		thrust::make_transform_iterator( /* input summands (scaled) */
			buf.begin(), thrust::placeholders::_1 / _size ),

		thrust::make_discard_iterator(), _mean.begin() ); /* output keys, means */

}

