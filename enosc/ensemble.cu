#include "hip/hip_runtime.h"
/**
 * ensemble.cu
 * 20150703
 *
 * ensemble interface
 */

	/* includes */
#include "ensemble.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* con/destruction */
enosc::Ensemble::Ensemble()
{

		/* initialize configuration */
	_size = 0;
	_dim = 0;

}

	/* configuration */
void enosc::Ensemble::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/size";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _size );

	std::string paramname = groupname + "/epsilon";
	std::string stepname = groupname + "/epsilon_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_epsilons.resize( steps+1 );
		_epsilons[0] = start;
		for ( unsigned int i = 1; i < steps+1; ++i )
			_epsilons[i] = start + i * (stop-start) / (enosc::scalar) steps;

	}

	paramname = groupname + "/beta";
	stepname = groupname + "/beta_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_betas.resize( steps+1 );
		_betas[0] = start;
		for ( unsigned int i = 1; i < steps+1; ++i )
			_betas[i] = start + i * (stop-start) / (enosc::scalar) steps;

	}

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "size: " << _size << "\n";
	logger.log() << "dim: " << _dim << "\n";

	logger.log() << "epsilons: " << _epsilons << "\n";
	logger.log() << "betas: " << _betas << "\n";

}

	/* phase space */
void enosc::Ensemble::init( unsigned int seed, bool det, bool stoch )
{

		/* prepare phase space buffers */
	_state.resize( _dim * _size * _epsilons.size() * _betas.size() );

	if ( det )
		_deriv_det.resize( _state.size() );
	if ( stoch )
		_deriv_stoch.resize( _state.size() );

		/* initialize randomness */
	srand( seed );

}

