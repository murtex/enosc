#include "hip/hip_runtime.h"
/**
 * observer.cu
 * 20150704
 *
 * observer interface
 */

	/* includes */
#include "observer.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* con/destruction */
enosc::Observer::Observer()
{

		/* default configuration */
	_meanfield = true;

	_raw = true;
	_raw_deriv = true;

	_polar = true;
	_polar_deriv = true;

}

enosc::Observer::~Observer()
{
}

	/* configuration */
void enosc::Observer::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/oscillators";
	if ( config.exists( settingname ) ) {
		unsigned int n = config.lookup( settingname ).getLength();

		_oscillators.resize( n );
		for ( unsigned int i = 0; i < n; ++i )
			_oscillators[i] = config.lookup( settingname )[i];
	}

	settingname = groupname + "/meanfield";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _meanfield );

	settingname = groupname + "/raw";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _raw );

	settingname = groupname + "/raw_deriv";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _raw_deriv );

	settingname = groupname + "/polar";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _polar );

	settingname = groupname + "/polar_deriv";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _polar_deriv );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "oscillators: " << _oscillators << "\n";
	logger.log() << "meanfield: " << _meanfield << "\n";

	logger.log() << "raw: " << _raw << "\n";
	logger.log() << "raw_deriv: " << _raw_deriv << "\n";

	logger.log() << "polar: " << _polar << "\n";
	logger.log() << "polar_deriv: " << _polar_deriv << "\n";

}

	/* observation */
void enosc::Observer::init( enosc::Ensemble const & ensemble, enosc::Stepper const & stepper, std::string const & filename )
{

		/* safeguard */
	for ( std::vector< unsigned int >::iterator it = _oscillators.begin(); it != _oscillators.end(); ++it )
		if ( (*it)-1 >= ensemble.get_size() )
			throw std::runtime_error( "invalid value: enosc::Observer::init, _oscillators" );

}


