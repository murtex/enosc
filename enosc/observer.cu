#include "hip/hip_runtime.h"
/**
 * observer.cu
 * 20150704
 *
 * observer interface
 */

	/* includes */
#include "observer.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* con/destruction */
enosc::Observer::Observer()
{
}

enosc::Observer::~Observer()
{
}

	/* configuration */
void enosc::Observer::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/oscillators";
	if ( config.exists( settingname ) ) {
	}

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "oscillators: " << _oscillators << "\n";

}

