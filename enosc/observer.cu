#include "hip/hip_runtime.h"
/**
 * observer.cu
 * 20150704
 *
 * observer interface
 */

	/* includes */
#include "observer.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* con/destruction */
enosc::Observer::Observer()
{

		/* default configuration */
	_size = 1;
	_meanfield = true;

}

enosc::Observer::~Observer()
{
}

	/* configuration */
void enosc::Observer::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/size";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _size );

	settingname = groupname + "/meanfield";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _meanfield );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "size: " << _size << "\n";
	logger.log() << "meanfield: " << _meanfield << "\n";

}

	/* observation */
void enosc::Observer::init( enosc::Ensemble const & ensemble, enosc::Stepper const & stepper, std::string const & filename )
{

		/* safeguard */
	if ( _size > ensemble.get_size() )
		throw std::runtime_error( "invalid value: enosc::Observer::init, _size" );

}


