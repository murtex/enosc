#include "hip/hip_runtime.h"
/**
 * observer.cu
 * 20150704
 *
 * observer interface
 */

	/* includes */
#include "observer.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* con/destruction */
enosc::Observer::Observer()
{

		/* default configuration */
	_transition = 0;

	_size = 1;
	_meanfield = true;

}

enosc::Observer::~Observer()
{
}

	/* configuration */
void enosc::Observer::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/transition_steps";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _transition );

	settingname = groupname + "/size";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _size );

	settingname = groupname + "/meanfield";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _meanfield );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "transition: " << _transition << "\n";

	logger.log() << "size: " << _size << "\n";
	logger.log() << "meanfield: " << _meanfield << "\n";

}

	/* observation */
void enosc::Observer::init( enosc::Ensemble const & ensemble, enosc::Stepper const & stepper, std::string const & filename )
{

		/* safeguard */
	if ( _transition > stepper.get_times().size()-1 )
		throw std::runtime_error( "invalid value: enosc::Observer::init, _transition" );

	if ( _size > ensemble.get_size() )
		throw std::runtime_error( "invalid value: enosc::Observer::init, _size" );

}


