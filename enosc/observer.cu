#include "hip/hip_runtime.h"
/**
 * observer.cu
 * 20150704
 *
 * observer interface
 */

	/* includes */
#include "observer.h"

#include <xis/singleton.h>
#include <xis/logger.h>

#include <enosc/kernels.h>

	/* con/destruction */
enosc::Observer::Observer()
{

		/* default configuration */
	_transition = 0;

	_size = 1;
	_meanfield = true;

	_track_raw = true;
	_track_polar = true;
	_track_funnel = true;

}

enosc::Observer::~Observer()
{
}

	/* configuration */
void enosc::Observer::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* parse group settings */
	std::string settingname = groupname + "/transition_steps";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _transition );

	settingname = groupname + "/size";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _size );

	settingname = groupname + "/mean";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _mean );

	settingname = groupname + "/meanfield";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _meanfield );

	settingname = groupname + "/track_raw";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _track_raw );

	settingname = groupname + "/track_polar";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _track_polar );

	settingname = groupname + "/track_funnel";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _track_funnel );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "transition: " << _transition << "\n";

	logger.log() << "size: " << _size << "\n";
	logger.log() << "mean: " << _mean << "\n";
	logger.log() << "meanfield: " << _meanfield << "\n";

	logger.log() << "track_raw: " << _track_raw << "\n";
	logger.log() << "track_polar: " << _track_polar << "\n";
	logger.log() << "track_funnel: " << _track_funnel << "\n";

}

	/* observation */
void enosc::Observer::init( enosc::Ensemble const & ensemble, enosc::Stepper const & stepper, std::string const & filename )
{

		/* safeguard */
	if ( _transition > stepper.get_times().size()-1 )
		throw std::runtime_error( "invalid value (enosc::Observer::init): _transition" );

	if ( _size > ensemble.get_size() )
		throw std::runtime_error( "invalid value (enosc::Observer::init): _size" );

		/* prepare buffers */
	_funnel.resize( ensemble.get_epsilons().size() * ensemble.get_betas().size() ); /* funneling */

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	size_t cuda_free;
	size_t cuda_total;
	hipMemGetInfo( &cuda_free, &cuda_total );
	logger.log() << "cuda: " << ((cuda_total-cuda_free)>>20) << "/" << (cuda_total>>20) << "MiB\n";

}

void enosc::Observer::compute_funnel( enosc::device_vector const & polar_deriv, unsigned int size )
{

		/* safeguard */
	if ( polar_deriv.size() % _funnel.size() != 0 )
		throw std::runtime_error( "invalid argument (enosc::Observer::compute_funnel): polar_deriv" );

		/* get minimum frequencies */
	unsigned int stride = _funnel.size();

	thrust::reduce_by_key(

		thrust::make_transform_iterator( /* input keys */
			thrust::counting_iterator< unsigned int >( 0 ),
			thrust::placeholders::_1 / size ),
		thrust::make_transform_iterator(
			thrust::counting_iterator< unsigned int >( 0 ),
			thrust::placeholders::_1 / size ) + stride * size,

		polar_deriv.begin() + stride * size, /* input frequencies */

		thrust::make_discard_iterator(), _funnel.begin(), /* output keys, minima */
		
		thrust::equal_to< unsigned int >(), thrust::minimum< enosc::scalar >() );

		/* zero positive frequencies */
	thrust::transform_if(
		_funnel.begin(), _funnel.begin() + stride * size,
		_funnel.begin(),
		enosc::SetZero(), enosc::IsPositive() );

}

