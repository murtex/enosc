#include "hip/hip_runtime.h"
/**
 * stepper.cu
 * 20150706
 *
 * stepper interface
 */

	/* includes */
#include "stepper.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* con/destruction */
enosc::Stepper::Stepper()
{

		/* initialize configuration */
	_dt = 0;

}

enosc::Stepper::~Stepper()
{

		/* release objects */
	delete _rnd;

}

	/* configuration */
void enosc::Stepper::configure( libconfig::Config const & config, std::string const & groupname )
{

	std::string paramname = groupname + "/time";
	std::string stepname = groupname + "/time_steps";
	if ( config.exists( paramname ) && config.exists( stepname ) ) {

			/* read settings */
		enosc::scalar start = config.lookup( paramname )[0];
		enosc::scalar stop = config.lookup( paramname )[1];

		unsigned int steps = config.lookup( stepname );

			/* set parameter values */
		_times.resize( steps+1 );
		_times[0] = start;
		for ( unsigned int i = 1; i < steps; ++i )
			_times[i] = start + i * (stop-start) / (enosc::scalar) steps;
		if ( steps >= 1 )
			_times[steps] = stop;

		if ( _times.size() > 1 )
			_dt = _times[1] - _times[0];

	}

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "times: " << _times << "\n";

}

	/* integration */
void enosc::Stepper::init( enosc::Ensemble const & ensemble )
{

		/* prepare randomness */
	_rng.seed( ensemble.get_seed() );
	_rnd = new std::normal_distribution< enosc::scalar >( 0, sqrt( _dt ) );

	_hrandom.resize( ensemble.get_dim() * ensemble.get_size() );
	_drandom.resize( ensemble.get_dim() * ensemble.get_epsilons().size() * ensemble.get_betas().size() * ensemble.get_size() );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	size_t cuda_free;
	size_t cuda_total;
	hipMemGetInfo( &cuda_free, &cuda_total );
	logger.log() << "cuda: " << ((cuda_total-cuda_free)>>20) << "/" << (cuda_total>>20) << "MiB\n";

}

