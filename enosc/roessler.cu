#include "hip/hip_runtime.h"
/**
 * roessler.cu
 * 20150703
 *
 * roessler ensemble
 */

	/* includes */
#include "roessler.h"

#include <xis/singleton.h>
#include <xis/logger.h>

#include <enosc/kernels.h>

	/* con/destruction */
enosc::Roessler::Roessler()
{

		/* initialize configuration */
	_dim = 3;

	_a = 0.15;
	_b = 0.4;
	_c = 8.5;

}

	/* configuration */
void enosc::Roessler::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* base call */
	enosc::Ensemble::configure( config, groupname );

		/* parse group settings */
	std::string settingname = groupname + "/a";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _a );

	settingname = groupname + "/b";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _b );

	settingname = groupname + "/c";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _c );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "a: " << _a << "\n";
	logger.log() << "b: " << _b << "\n";
	logger.log() << "c: " << _c << "\n";

}

	/* phase space */
void enosc::Roessler::init()
{

		/* base call */
	enosc::Ensemble::init();

		/* prepare random state */
	enosc::host_vector rs( _size );
	enosc::host_vector phis( _size );
	enosc::host_vector zs( _size );

	for ( unsigned int i = 0; i < _size; ++i ) {
		rs[i] = rand() / (double) RAND_MAX * 5 + 7.5; /* [7.5..12.5] */
		phis[i] = rand() / (double) (RAND_MAX-1) * 2*M_PI; /* [0..2pi) */
		zs[i] = rand() / (double) RAND_MAX * 0.5; /* [0..0.5] */
	}

		/* transfer random state */
	enosc::device_vector drs( rs );
	enosc::device_vector dphis( phis );
	enosc::device_vector dzs( zs );

	thrust::for_each_n(
		thrust::make_zip_iterator( thrust::make_tuple(

			thrust::make_permutation_iterator( /* cylinder input */
				drs.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 % _size ) ),
			thrust::make_permutation_iterator(
				dphis.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 % _size ) ),
			thrust::make_permutation_iterator(
				dzs.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 % _size ) ),

			_state.begin(), /* cartesian output */
			_state.begin() + _epsilons.size() * _betas.size() * _size,
			_state.begin() + 2 * _epsilons.size() * _betas.size() * _size ) ),

		_epsilons.size() * _betas.size() * _size, enosc::CylinderToCartesian() );

}

	/* ode */
bool enosc::Roessler::compute_deriv_det( enosc::device_vector const & state, enosc::scalar time )
{

		/* safeguard */
	if ( state.size() != _state.size() )
		throw std::runtime_error( "invalid argument: enosc::Roessler::compute_deriv_det, state" );

		/* compute ode */
	compute_mean( state );

	thrust::for_each_n(
		thrust::make_zip_iterator( thrust::make_tuple(

			state.begin(), /* state input */
			state.begin() + _epsilons.size() * _betas.size() * _size,
			state.begin() + 2 * _epsilons.size() * _betas.size() * _size,

			thrust::make_permutation_iterator( /* coupling input */
				_epsilons.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					(thrust::placeholders::_1 / (_betas.size() * _size)) % _epsilons.size() ) ),
			thrust::make_permutation_iterator(
				_betas.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
                    (thrust::placeholders::_1 / _size) % _betas.size() ) ),

            thrust::make_permutation_iterator( /* meanfield input */
				_mean.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 / _size ) ),
			thrust::make_permutation_iterator(
				_mean.begin() + _epsilons.size() * _betas.size(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 / _size ) ),

			_deriv.begin(), /* derivative output */
			_deriv.begin() + _epsilons.size() * _betas.size() * _size,
			_deriv.begin() + 2 * _epsilons.size() * _betas.size() * _size ) ),

		_epsilons.size() * _betas.size() * _size, enosc::RoesslerODE( _a, _b, _c ) );

	return true;
}

