#include "hip/hip_runtime.h"
/**
 * roessler.cu
 * 20150703
 *
 * roessler ensemble
 */

	/* includes */
#include "roessler.h"

#include <xis/singleton.h>
#include <xis/logger.h>

#include <enosc/transforms.h>

	/* con/destruction */
enosc::Roessler::Roessler()
{

		/* initialize configuration */
	_dim = 3;

	_a = 0.15;
	_b = 0.4;
	_c = 8.5;

}

	/* configuration */
void enosc::Roessler::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* base call */
	enosc::Ensemble::configure( config, groupname );

		/* parse group settings */
	std::string settingname = groupname + "/a";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _a );

	settingname = groupname + "/b";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _b );

	settingname = groupname + "/c";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _c );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "a: " << _a << "\n";
	logger.log() << "b: " << _b << "\n";
	logger.log() << "c: " << _c << "\n";

}

	/* phase space */
void enosc::Roessler::init( unsigned int seed, bool det, bool stoch )
{

		/* base call */
	enosc::Ensemble::init( seed, true, false );

		/* prepare random state */
	enosc::host_vector rs( _size );
	enosc::host_vector phis( _size );
	enosc::host_vector zs( _size );

	for ( unsigned int i = 0; i < _size; ++i ) {
		rs[i] = rand() / (double) RAND_MAX * 5 + 7.5; /* [7.5..12.5] */
		phis[i] = rand() / (double) (RAND_MAX-1) * 2*M_PI; /* [0..2pi) */
		zs[i] = rand() / (double) RAND_MAX * 0.5; /* [0..0.5] */
	}

		/* transfer random state */
	enosc::device_vector drs( rs );
	enosc::device_vector dphis( phis );
	enosc::device_vector dzs( zs );

	thrust::for_each_n(
		thrust::make_zip_iterator( thrust::make_tuple(

			thrust::make_permutation_iterator( /* cylinder input */
				drs.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 % _size ) ),
			thrust::make_permutation_iterator(
				dphis.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 % _size ) ),
			thrust::make_permutation_iterator(
				dzs.begin(),
				thrust::make_transform_iterator(
					thrust::counting_iterator< unsigned int >( 0 ),
					thrust::placeholders::_1 % _size ) ),

			_state.begin(), /* cartesian output */
			_state.begin() + _size * _epsilons.size() * _betas.size(),
			_state.begin() + 2 * _size * _epsilons.size() * _betas.size() ) ),

		_size * _epsilons.size() * _betas.size(), enosc::CylinderToCartesian() );

}

enosc::device_vector const & enosc::Roessler::compute_deriv_det( enosc::device_vector const & state, enosc::scalar time )
{

	return _deriv_det;
}

