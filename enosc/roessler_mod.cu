#include "hip/hip_runtime.h"
/**
 * roessler_mod.cu
 * 20150704
 *
 * modified roessler ensemble
 */

	/* includes */
#include "roessler_mod.h"

#include <xis/singleton.h>
#include <xis/logger.h>

	/* interface */
enosc::RoesslerMod::RoesslerMod()
{

		/* initialize configuration */
	_c = 10.0;

	_e = 0.15;
	_f = 0.2;

}

	/* configuration */
void enosc::RoesslerMod::configure( libconfig::Config const & config, std::string const & groupname )
{

		/* base call */
	enosc::Roessler::configure( config, groupname );

		/* parse group settings */
	std::string settingname = groupname + "/e";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _e );

	settingname = groupname + "/f";
	if ( config.exists( settingname ) )
		config.lookupValue( settingname, _f );

		/* logging */
	xis::Logger & logger = xis::Singleton< xis::Logger >::instance();

	logger.log() << "e: " << _e << "\n";
	logger.log() << "f: " << _f << "\n";

}

