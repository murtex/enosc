#include "hip/hip_runtime.h"
/**
 * integrate.cu
 * 20150701
 *
 * integrate dynamical system
 */

	/* includes */
#include <unistd.h>

#include <xis/singleton.h>
#include <xis/logger.h>

#include <xis/factory.h>
#include <enosc/ensemble.h>
#include <enosc/roessler.h>
#include <enosc/roessler_mod.h>
#include <enosc/sakaguchi.h>
#include <enosc/sakaguchi_add.h>

#include <enosc/observer.h>
#include <enosc/hdf5.h>

#include <enosc/stepper.h>
#include <enosc/euler.h>
#include <enosc/heun.h>

	/* command line */
char const * _cl_config = NULL;
char const * _cl_include = NULL;
char const * _cl_output = NULL;

void cl_usage( char * cmd )
{
	std::cout << "usage: " << cmd << " [-h] -c config [-i include] [-o output]" << std::endl;
}

void cl_parse( int argc, char ** argv )
{

		/* proceed command line options */
	int optc;

	while ( (optc = getopt( argc, argv, "hc:i:o:" ) ) != -1 )
		switch ( optc ) {

			case '?': /* general */
				cl_usage( argv[0] );
				exit( 1 );
			case 'h':
				cl_usage( argv[0] );
				exit( 0 );

			case 'c': /* arguments */
				_cl_config = optarg;
				break;
			case 'i':
				_cl_include = optarg;
				break;
			case 'o':
				_cl_output = optarg;
				break;

		}

		/* check options */
	if ( _cl_config == NULL )
		throw std::runtime_error( "invalid value (cl_parse): _cl_config" );

	if ( _cl_output == NULL )
		throw std::runtime_error( "invalid value (cl_parse): _cl_output" );

}

	/* workflow */
libconfig::Config & _config = xis::Singleton< libconfig::Config >::instance();

xis::Logger & _logger = xis::Singleton< xis::Logger >::instance();

enosc::Ensemble * _ensemble = NULL;
enosc::Stepper * _stepper = NULL;
enosc::Observer * _observer = NULL;

void init()
{

		/* read configuration file */
	if ( _cl_include != NULL )
		_config.setIncludeDir( _cl_include );

	_config.readFile( _cl_config );

		/* configure logger */
	_logger.configure( _config, "logger" );

		/* initialize ensemble */
	xis::Factory< enosc::Ensemble > fac_ensemble; /* registration */

	fac_ensemble.enroll< enosc::Roessler >( "roessler" );
	fac_ensemble.enroll< enosc::RoesslerMod >( "roessler-mod" );
	fac_ensemble.enroll< enosc::Sakaguchi >( "sakaguchi" );
	fac_ensemble.enroll< enosc::SakaguchiAdd >( "sakaguchi-add" );

	std::string ensemble = _config.lookup( "ensemble/type" ); /* creation */
	_logger.tab() << "create ensemble (" << ensemble << ")...\n";

	_ensemble = fac_ensemble.create( ensemble );
	_ensemble->configure( _config, "ensemble" );
	_ensemble->init();

	_logger.untab();

		/* initialize stepper */
	xis::Factory< enosc::Stepper > fac_stepper; /* registration */

	fac_stepper.enroll< enosc::Euler >( "euler" );
	fac_stepper.enroll< enosc::Heun >( "heun" );

	std::string stepper = _config.lookup( "stepper/type" ); /* creation */
	_logger.tab() << "create stepper (" << stepper << ")...\n";

	_stepper = fac_stepper.create( stepper );
	_stepper->configure( _config, "stepper" );
	_stepper->init( *_ensemble );

	_logger.untab();

		/* initialize observer */
	xis::Factory< enosc::Observer > fac_observer; /* registration */

	fac_observer.enroll< enosc::HDF5 >( "hdf5" );

	std::string observer = _config.lookup( "observer/type" ); /* creation */
	_logger.tab() << "create observer (" << observer << ")...\n";

	_observer = fac_observer.create( observer );
	_observer->configure( _config, "observer" );
	_observer->init( *_ensemble, *_stepper, _cl_output );

	_logger.untab();

}

void exit()
{
}

void run()
{

		/* integrate and observe ensemble */
	enosc::host_vector const & times = _stepper->get_times();
	unsigned int steps = times.size() - 1;

	if ( steps == 0 ) /* initial state only */
		_observer->observe( *_ensemble, 0, times[0] );

	else { /* continuous integration */
		_logger.progress( steps, steps ) << "integrate ensemble...\n";
		for ( unsigned int i = 0; i < steps; ++i ) {
			_logger.progress( i, steps );

			if ( i == 0 ) /* initial state */
				_observer->observe( *_ensemble, i, times[i] );

			_stepper->integrate( *_ensemble, i ); /* integrated state */
			_observer->observe( *_ensemble, i+1, times[i+1] );

			_ensemble->swap(); /* swap buffers */

		}
	}

}

	/* main */
int main( int argc, char ** argv )
{

		/* parse command line */
	cl_parse( argc, argv );

		/* proceed workflow */
	init();
	run();
	exit();

	_logger.log() << "done.\n";

	return 0;
}

