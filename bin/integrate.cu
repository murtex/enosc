#include "hip/hip_runtime.h"
/**
 * integrate.cu
 * 20150701
 *
 * integrate dynamical system
 */

	/* includes */
#include <unistd.h>

#include <xis/singleton.h>
#include <xis/logger.h>

#include <xis/factory.h>
#include <enosc/ensemble.h>
#include <enosc/roessler.h>

#include <enosc/observer.h>
#include <enosc/hdf5.h>

#include <enosc/stepper.h>
#include <enosc/euler.h>

	/* command line */
char const * _cl_config = NULL;
char const * _cl_include = NULL;
char const * _cl_output = NULL;

void cl_usage( char * cmd )
{
	std::cout << "usage: " << cmd << " [-h] [-c config] [-i include] [-o output]" << std::endl;
}

void cl_parse( int argc, char ** argv )
{

		/* proceed command line options */
	int optc;

	while ( (optc = getopt( argc, argv, "hc:i:o:" ) ) != -1 )
		switch ( optc ) {

			case '?': /* general */
				cl_usage( argv[0] );
				exit( 1 );
			case 'h':
				cl_usage( argv[0] );
				exit( 0 );

			case 'c': /* arguments */
				_cl_config = optarg;
				break;
			case 'i':
				_cl_include = optarg;
				break;
			case 'o':
				_cl_output = optarg;
				break;

		}

}

	/* workflow */
libconfig::Config & _config = xis::Singleton< libconfig::Config >::instance();

xis::Logger & _logger = xis::Singleton< xis::Logger >::instance();

enosc::Ensemble * _ensemble = NULL;
enosc::Stepper * _stepper = NULL;
enosc::Observer * _observer = NULL;

void init()
{

		/* read configuration file */
	if ( _cl_include != NULL )
		_config.setIncludeDir( _cl_include );

	if ( _cl_config != NULL )
		_config.readFile( _cl_config );

		/* configure logger */
	_logger.configure( _config, "logger" );

		/* initialize ensemble */
	xis::Factory< enosc::Ensemble > fac_ensemble; /* registration */

	fac_ensemble.enroll< enosc::Roessler >( "roessler" );

	std::string ensemble = _config.lookup( "ensemble/type" ); /* creation */
	_logger.tab() << "create ensemble (" << ensemble << ")...\n";

	_ensemble = fac_ensemble.create( ensemble );
	_ensemble->configure( _config, "ensemble" );
	_ensemble->init();

	_logger.untab();

		/* initialize stepper */
	xis::Factory< enosc::Stepper > fac_stepper; /* registration */

	fac_stepper.enroll< enosc::Euler >( "euler" );

	std::string stepper = _config.lookup( "stepper/type" ); /* creation */
	_logger.tab() << "create stepper (" << stepper << ")...\n";

	_stepper = fac_stepper.create( stepper );
	_stepper->configure( _config, "stepper" );

	_logger.untab();

		/* initialize observer */
	xis::Factory< enosc::Observer > fac_observer; /* registration */

	fac_observer.enroll< enosc::HDF5 >( "hdf5" );

	std::string observer = _config.lookup( "observer/type" ); /* creation */
	_logger.tab() << "create observer (" << observer << ")...\n";

	_observer = fac_observer.create( observer );
	_observer->configure( _config, "observer" );
	_observer->init( *_ensemble, *_stepper, _cl_output );

	_logger.untab();

}

void exit()
{
}

void run()
{

		/* integrate ensemble */
	unsigned int steps = _stepper->get_times().size() - 1;

	if ( steps > 0 ) {

		_logger.progress() << "integrate ensemble...\n";
		for ( unsigned int i = 0; i < steps; ++i ) {
			_logger.progress( i, steps );

			_stepper->integrate( *_ensemble, _stepper->get_times()[i] );

			_observer->observe( *_ensemble, _stepper->get_times()[i] );

		}

	}

}

	/* main */
int main( int argc, char ** argv )
{

		/* parse command line */
	cl_parse( argc, argv );

		/* proceed workflow */
	init();
	run();
	exit();

	_logger.log() << "done.\n";

	return 0;
}

